#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <torch/script.h>
#include <torch/torch.h>

#include <cstdint>
#include <iostream>
#include <vector>

#include "ATen/Functions.h"
#include "ATen/core/TensorBody.h"
#include "segment_mean_cuda.h"

using torch::autograd::AutogradContext;
using torch::autograd::Variable;
using torch::autograd::variable_list;

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS


template <typename scalar_t>
__global__ void segment_mean_cuda_forward_kernel(
    const scalar_t *x_data, const int64_t *index_data, scalar_t *out_data,
    scalar_t *count_data, int64_t E, int64_t K, int64_t N, int64_t numel) {
  int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t e = (thread_idx / K) % E;
  int64_t k = thread_idx % K;
  if (thread_idx < numel) {
    // TODO: support more data type
    int64_t idx = index_data[e];
    atomicAdd(out_data + idx * K + k, x_data[thread_idx]);
    atomicAdd(count_data + idx * K + k, 1.);
  }
}

// TODO: fuse segment & arg_segment to one kernel function.
template <typename scalar_t>
__global__ void arg_segment_mean_cuda_forward_kernel(
    const scalar_t *x_data, const int64_t *index_data, scalar_t *out_data,
    scalar_t *count_data, int64_t E, int64_t K,
    int64_t N, int64_t numel) {
  int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel) {
    if (count_data[thread_idx] > 0) {
      out_data[thread_idx] /= count_data[thread_idx];
    }
  }
}

torch::Tensor segment_mean_cuda_forward(
    torch::Tensor x, torch::Tensor index, int64_t N) {
  // check inputs
  TORCH_CHECK(x.device().is_cuda(), "x must be CUDA tensor");
  TORCH_CHECK(index.device().is_cuda(), "index must be CUDA tensor");
  TORCH_CHECK_INDEX(
      index.dim() == 1, "index dimension should be 1, but got ", index.dim());
  TORCH_CHECK_INDEX(
      x.size(0) == index.size(0),
      "fisrt dimension of x and index should be same");
  // only support float Tensor
  // TORCH_CHECK_TYPE(
  //     x.scalar_type() == c10::ScalarType::Float, "x should be float Tensor")
  hipSetDevice(x.get_device());
  x = x.contiguous();
  index = index.contiguous();

  auto sizes = x.sizes().vec();
  sizes[0] = N > *index.max().cpu().data_ptr<int64_t>()
                 ? N
                 : *index.max().cpu().data_ptr<int64_t>();
  torch::Tensor out = torch::zeros(sizes, x.options());
  // TORCH_CHECK(out.device().is_cuda(), "out must be CUDA tensor");
  if (x.numel() == 0) {
    return out;
  }

  auto E = x.size(0);
  auto K = x.numel() / x.size(0);
  auto stream = at::cuda::getCurrentCUDAStream();

  if (x.dtype() == torch::kInt8 || x.dtype() == torch::kInt16 || x.dtype() == torch::kInt32 || x.dtype() == torch::kInt64) {
    auto type = x.dtype();
    using scalar_t = int;
    if (x.dtype() == torch::kInt8 || x.dtype() == torch::kInt16 || x.dtype() == torch::kInt64) {
      x = x.to(torch::kInt32);
      out = out.to(torch::kInt32);
    }
    // using scalar_t = float;  // temporary usage, delete later
    auto x_data = x.data_ptr<scalar_t>();
    auto out_data = out.data_ptr<scalar_t>();
    auto index_data = index.data_ptr<int64_t>();

    torch::Tensor count = torch::full_like(out, 0.0, x.options());
    scalar_t *count_data = count.data_ptr<scalar_t>();

    segment_mean_cuda_forward_kernel<scalar_t>
        <<<BLOCKS(x.numel()), THREADS, 0, stream>>>(
            x_data, index_data, out_data, count_data, E, K, N, x.numel());

    arg_segment_mean_cuda_forward_kernel<scalar_t>
        <<<BLOCKS(out.numel()), THREADS, 0, stream>>>(
            x_data, index_data, out_data, count_data, E, K, out.sizes().vec()[0],
            out.numel());
    
    out = out.to(type);
  } else if (x.dtype() == torch::kFloat16 || x.dtype() == torch::kFloat32) {
    auto type = x.dtype();
    using scalar_t = float;
    if (x.dtype() == torch::kFloat16) {
      x = x.to(torch::kFloat32);
      out = out.to(torch::kFloat32);
    }

    auto x_data = x.data_ptr<scalar_t>();
    auto out_data = out.data_ptr<scalar_t>();
    auto index_data = index.data_ptr<int64_t>();

    torch::Tensor count = torch::full_like(out, 0.0, x.options());
    scalar_t *count_data = count.data_ptr<scalar_t>();

    segment_mean_cuda_forward_kernel<scalar_t>
        <<<BLOCKS(x.numel()), THREADS, 0, stream>>>(
            x_data, index_data, out_data, count_data, E, K, N, x.numel());

    arg_segment_mean_cuda_forward_kernel<scalar_t>
        <<<BLOCKS(out.numel()), THREADS, 0, stream>>>(
            x_data, index_data, out_data, count_data, E, K, N,
            out.numel());
    
    out = out.to(type);
  } else if (x.dtype() == torch::kFloat64) {
    using scalar_t = double;
    auto x_data = x.data_ptr<scalar_t>();
    auto out_data = out.data_ptr<scalar_t>();
    auto index_data = index.data_ptr<int64_t>();

    torch::Tensor count = torch::full_like(out, 0.0, x.options());
    scalar_t *count_data = count.data_ptr<scalar_t>();

    segment_mean_cuda_forward_kernel<scalar_t>
        <<<BLOCKS(x.numel()), THREADS, 0, stream>>>(
            x_data, index_data, out_data, count_data, E, K, N, x.numel());

    arg_segment_mean_cuda_forward_kernel<scalar_t>
        <<<BLOCKS(out.numel()), THREADS, 0, stream>>>(
            x_data, index_data, out_data, count_data, E, K, N,
            out.numel());
  }

  return out;
}
