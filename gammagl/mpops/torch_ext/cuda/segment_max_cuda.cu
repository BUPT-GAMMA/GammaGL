#include "hip/hip_runtime.h"
#include "segment_max_cuda.h"
#include <torch/torch.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <torch/script.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>

using torch::autograd::AutogradContext;
using torch::autograd::Variable;
using torch::autograd::variable_list;

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

inline __device__ void atomic_max_float(float *addr, float value) {
  int *addr_as_i = (int *)addr;
  int old = *addr_as_i;
  int assumed;
  do{
    assumed = old;
    old = atomicCAS(addr_as_i, assumed,
                    __float_as_int(max(value, __int_as_float(assumed))));
  } while (assumed != old);
}

template <typename scalar_t>
__global__ void segment_max_cuda_forward_kernel(const scalar_t *x_data, const int64_t *index_data,
                               scalar_t *out_data, int64_t E, int64_t K, int64_t N, int64_t numel) {
  int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t e = (thread_idx / K) % E;
  int64_t k = thread_idx % K;
  if (thread_idx < numel)  {
    // TODO: support more data type
    int64_t idx = index_data[e];
    atomic_max_float(out_data + idx * K + k,
                     x_data[thread_idx]);
  }
}

// TODO: fuse segment & arg_segment to one kernel function.
template <typename scalar_t>
__global__ void
arg_segment_max_cuda_forward_kernel(const scalar_t *x_data, const int64_t *index_data,
                   scalar_t *out_data, int64_t *arg_out_data, int64_t E,
                   int64_t K, int64_t N, int64_t numel) {
  int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t e = (thread_idx / K) % E;
  int64_t k = thread_idx % K;

  if (thread_idx < numel) {
    int64_t idx = index_data[e];
    if (x_data[thread_idx] == out_data[idx * K + k]) {
      arg_out_data[idx * K + k] = e;
    }
  }
}

std::tuple<torch::Tensor, torch::Tensor>
segment_max_cuda_forward(torch::Tensor x, torch::Tensor index, int64_t N) {
  // check inputs
  TORCH_CHECK(x.device().is_cuda(), "x must be CUDA tensor");
  TORCH_CHECK(index.device().is_cuda(), "index must be CUDA tensor");
  TORCH_CHECK_INDEX(x.dim() == 2, "x dimension should be 2, but got ", x.dim());
  TORCH_CHECK_INDEX(index.dim() == 1, "index dimension should be 1, but got ", index.dim());
  TORCH_CHECK_INDEX(x.size(0) == index.size(0), "fisrt dimension of x and index should be same");
  // only support float Tensor
  TORCH_CHECK_TYPE(x.scalar_type() == c10::ScalarType::Float, "x should be float Tensor")
  hipSetDevice(x.get_device());
  x = x.contiguous();

  auto sizes = x.sizes().vec();
  sizes[0] = N > *index.max().cpu().data_ptr<int64_t>()
                 ? N
                 : *index.max().cpu().data_ptr<int64_t>();
  torch::Tensor out = torch::empty(sizes, x.options());
  // TORCH_CHECK(out.device().is_cuda(), "out must be CUDA tensor");
  torch::Tensor arg_out = torch::full_like(out, 0, index.options());
  int64_t *arg_out_data = arg_out.data_ptr<int64_t>();
  if (x.numel() == 0) {
    out.fill_(0);
    return std::make_tuple(out, arg_out);
  }

  out.fill_(std::numeric_limits<int64_t>::lowest());
  auto E = x.size(0);
  auto K = x.size(1);
  auto stream = at::cuda::getCurrentCUDAStream();

  // AT_DISPATCH_ALL_TYPES(x.scalar_type(), "__ops_name",  [&] {
  using scalar_t = float; // temporary usage, delete later
  auto x_data = x.data_ptr<scalar_t>();
  auto out_data = out.data_ptr<scalar_t>();
  auto index_data = index.data_ptr<int64_t>();

  segment_max_cuda_forward_kernel<scalar_t>
      <<<BLOCKS(x.numel()), THREADS, 0, stream>>>(
          x_data, index_data, out_data, E, K, N, x.numel());

  out.masked_fill_(out == std::numeric_limits<int64_t>::lowest(), (scalar_t)0);

  arg_segment_max_cuda_forward_kernel<scalar_t>
      <<<BLOCKS(x.numel()), THREADS, 0, stream>>>(
          x_data, index_data, out_data, arg_out_data, E, K, N,
          x.numel());
  // });

  return std::make_tuple(out, arg_out);
}
